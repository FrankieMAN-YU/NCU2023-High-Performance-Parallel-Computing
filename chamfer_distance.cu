#include <iostream>
#include <cmath>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <time.h>

#define THREADS_PER_BLOCK 2048
const int NodeNumber = 100000000;
using namespace std;

// CUDA Kernel函数，计算三维点云chamfer distance
__global__ void chamferDistanceCUDA(const float* points1, const float* points2, int numPoints1, int numPoints2, float* distances) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < numPoints1) {
        float minDist = INFINITY;
        float x1 = points1[tid * 3];
        float y1 = points1[tid * 3 + 1];
        float z1 = points1[tid * 3 + 2];
        for (int i = 0; i < numPoints2; ++i) {
            float x2 = points2[i * 3];
            float y2 = points2[i * 3 + 1];
            float z2 = points2[i * 3 + 2];
            float dx = x1 - x2;
            float dy = y1 - y2;
            float dz = z1 - z2;
            float dist = dx * dx + dy * dy + dz * dz;
            minDist = fminf(minDist, dist);
        }
        distances[tid] = minDist;
    }
}

// 计算三维点云chamfer distance的函数
void computeChamferDistance(const float* points1, const float* points2, int numPoints1, int numPoints2, float* distances) {
    // 将点云数据传输到设备内存
    float* d_points1;
    float* d_points2;
    float* d_distances;
    hipMalloc((void**)&d_points1, numPoints1 * 3 * sizeof(float));
    hipMalloc((void**)&d_points2, numPoints2 * 3 * sizeof(float));
    hipMalloc((void**)&d_distances, numPoints1 * sizeof(float));
    hipMemcpy(d_points1, points1, numPoints1 * 3 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_points2, points2, numPoints2 * 3 * sizeof(float), hipMemcpyHostToDevice);

    // 设置CUDA Grid和Block的大小
    dim3 blockDim(THREADS_PER_BLOCK);
    dim3 gridDim((numPoints1 + blockDim.x - 1) / blockDim.x);

    // 调用CUDA Kernel函数计算三维点云chamfer distance
    chamferDistanceCUDA<<<gridDim, blockDim>>>(d_points1, d_points2, numPoints1, numPoints2, d_distances);

    // 将结果从设备内存传输回主机内存
    hipMemcpy(distances, d_distances, numPoints1 * sizeof(float), hipMemcpyDeviceToHost);

    // 释放设备内存
    hipFree(d_points1);
    hipFree(d_points2);
    hipFree(d_distances);
}

int main() {
    // 定义点云数据和参数
    const int numPoints1 = NodeNumber;
    const int numPoints2 = NodeNumber;
    float* points1 = new float[numPoints1 * 3];
    float* points2 = new float[numPoints2 * 3];
    float* distances = new float[numPoints1];

    // 生成随机的点云数据（示例）
    for (int i = 0; i < numPoints1; ++i) {
        points1[i * 3] = static_cast<float>(rand()) / RAND_MAX;
        points1[i * 3 + 1] = static_cast<float>(rand()) / RAND_MAX;
        points1[i * 3 + 2] = static_cast<float>(rand()) / RAND_MAX;
    }
    for (int i = 0; i < numPoints2; ++i) {
        points2[i * 3] = static_cast<float>(rand()) / RAND_MAX;
        points2[i * 3 + 1] = static_cast<float>(rand()) / RAND_MAX;
        points2[i * 3 + 2] = static_cast<float>(rand()) / RAND_MAX;
    }

    // 调用函数计算点云chamfer distance
    struct timeval start, stop;
    gettimeofday(&start, NULL);
    computeChamferDistance(points1, points2, numPoints1, numPoints2, distances);
    gettimeofday(&stop, NULL);
    
    // 计算运算时间
    double elapse = (stop.tv_sec - start.tv_sec) * 1000 + (stop.tv_usec - start.tv_usec) / 1000;
    cout << "RTX 3060 CUDA runtime = " << elapse << "ms, number nodes = " << NodeNumber << ", thread per block = " << THREADS_PER_BLOCK << endl;

    // 打印计算结果
    //for (int i = 0; i < numPoints1; ++i) {
    //    std::cout << "Point " << i << ": " << distances[i] << std::endl;
    //}

    // 释放内存
    delete[] points1;
    delete[] points2;
    delete[] distances;

    return 0;
}
